#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void helloCUDA() {
    printf("Hello, CUDA World!\n");
}

int main() {
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the GPU to finish
    return 0;
}
